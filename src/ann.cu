#include "hip/hip_runtime.h"
#include "ann.h"

namespace ann {


	__global__ void
	kernel(int n, float *arr){

		volatile int idx = threadIdx.x + blockDim.x*blockIdx.x;
		if(idx >= n) return;

	        arr[idx] *= 2.0f;
	}

	__global__ void
	kernel_feedforward(
		int layer_id,
		int *l,
		int *s,
		int *sw,
		float *z_arr,
		float *a_arr,
		float *w_arr
	 ){
		volatile int idx = threadIdx.x + blockDim.x*blockIdx.x;

		int neuron_count = l[layer_id];
		int neuron_count_prev = l[layer_id-1];

		//printf("layer = %d idx = %d count = %d\n", layer_id, idx, neuron_count-1);
		if(idx >= neuron_count-1) return;

		float z = 0;
		for(int k = 0; k < neuron_count_prev; k++){
			z += w_arr[sw[layer_id-1] + k*(neuron_count - 1) + idx]*a_arr[s[layer_id-1] + k];
			// printf("w_arr[%d] * a_arr[%d] = %.20f\n",
			// 		sw[layer_id-1] + k*(neuron_count - 1) + idx ,
			// 		s[layer_id-1] + k,
			// 		w_arr[sw[layer_id-1] + k*(neuron_count - 1) + idx]*a_arr[s[layer_id-1] + k]);
			// printf("%.10f * %.10f = %.10f\n", w_arr[sw[layer_id-1] + k*(neuron_count - 1) + idx ],
			// 		a_arr[s[layer_id-1] + k],
			// 		w_arr[sw[layer_id-1] + k*(neuron_count - 1) + idx]*a_arr[s[layer_id-1] + k]
			// 	);

		}

		z_arr[s[layer_id] + idx] = z;
		float a = 1.0 / (1.0 + expf(-z));
		a_arr[s[layer_id] + idx] = a;
	  // printf("index = %d z = %.5f\n", s[layer_id] + idx, z);
		// printf("a = %.20f\n", a);
	}

	__global__ void
	kernel_calc_gL(
		int layer_id,
		int *l,
		int *s,
		float *z_arr,
		float *a_arr,
		float *t_arr,
		float *gjl
	 ){

		volatile int idx = threadIdx.x + blockDim.x*blockIdx.x;

		int neuron_count = l[layer_id];

		if(idx >= neuron_count-1) return;

		float z = z_arr[s[layer_id] + idx];
		float tmp = 1 + expf(-z);
		float f_deriv=expf(-z) / (tmp*tmp);

		gjl[s[layer_id] + idx] = f_deriv*(a_arr[s[layer_id] + idx] - t_arr[idx]);
	}

	__global__ void
	kernel_calc_gjL(
		int layer_id,
		int *l,
		int *s,
		int *sw,
		float *z_arr,
		float *a_arr,
		float *t_arr,
		float *gjl,
		float *w_arr
	 ){

		volatile int idx = threadIdx.x + blockDim.x*blockIdx.x;

		int neuron_count = l[layer_id];
		int neuron_count_next = l[layer_id+1];

		if(idx >= neuron_count-1) return;

		//float f_deriv=expf(-z_arr[s[layer_id] + idx]) / powf((1 + expf(-z_arr[s[layer_id] + idx])),2.0f);
		float z = z_arr[s[layer_id] + idx];
		float tmp = 1 + expf(-z);
		float f_deriv=expf(-z) / (tmp*tmp);


		float sum = 0;
		for (int k = 0; k < neuron_count_next-1; k++) {
				sum += w_arr[sw[layer_id] + idx*(l[layer_id + 1] - 1) + k] * gjl[s[layer_id + 1] + k];
		}

		gjl[s[layer_id] + idx] = f_deriv*sum;
		// printf("Kernelis %d - %.20f\n", s[layer_id] + idx, gjl[s[layer_id] + idx]);
	}


	__global__ void
	kernel_weight_update(
		int layer_id,
		int *l,
		int *s,
		int *sw,
		float *z_arr,
		float *a_arr,
		float *t_arr,
		float *gjl,
		float *w_arr,
		float *dw_arr,
		float eta,
		float alpha
	 ){

		 volatile int idx = threadIdx.x + blockDim.x*blockIdx.x;

		 int neuron_count = l[layer_id];
		 int neuron_count_next = l[layer_id+1];

		 if(idx >= neuron_count) return;

		 float a = a_arr[s[layer_id] + idx];
		 for(int k = 0; k < neuron_count_next-1; k++){

			 float grad=/*a_arr[s[layer_id] + idx]*/a*gjl[s[layer_id + 1] + k];

			 dw_arr[sw[layer_id] + idx*(neuron_count_next - 1) + k]=
			 		-eta*grad+
			 		alpha*dw_arr[sw[layer_id] + idx*(neuron_count_next - 1) + k];

			 w_arr[sw[layer_id] + idx*(neuron_count_next - 1) + k]+=
			 		dw_arr[sw[layer_id] + idx*(neuron_count_next - 1) + k];
		 }
	}

	// CUDA2
	__global__ void
	kernel_feedforward_2(
		int layer_id,
		int *l,
		int *s_ext,
		int *sw_ext,
		float *z_ext_arr,
		float *a_ext_arr,
		float *w_arr
	 ){
		volatile int idx = threadIdx.x + blockDim.x*blockIdx.x;

		int neuron_count = l[layer_id];
		int neuron_count_prev = l[layer_id-1];

		//printf("layer = %d idx = %d count = %d\n", layer_id, idx, neuron_count-1);
		if(idx >= neuron_count-1) return;

		float z = 0;
		for(int k = 0; k < neuron_count_prev; k++){
			z += w_arr[sw_ext[layer_id-1] + k*(neuron_count - 1) + idx]*a_ext_arr[s_ext[layer_id-1] + k];
			// printf("w_arr[%d] * a_arr[%d] = %.20f\n",
			// 		sw[layer_id-1] + k*(neuron_count - 1) + idx ,
			// 		s[layer_id-1] + k,
			// 		w_arr[sw[layer_id-1] + k*(neuron_count - 1) + idx]*a_arr[s[layer_id-1] + k]);
			// printf("%.10f * %.10f = %.10f\n", w_arr[sw[layer_id-1] + k*(neuron_count - 1) + idx ],
			// 		a_arr[s[layer_id-1] + k],
			// 		w_arr[sw[layer_id-1] + k*(neuron_count - 1) + idx]*a_arr[s[layer_id-1] + k]
			// 	);

		}

		z_ext_arr[s_ext[layer_id] + idx] = z;
		float a = 1.0 / (1.0 + expf(-z));
		a_ext_arr[s_ext[layer_id] + idx] = a;
		// printf("index = %d z = %.5f\n", s[layer_id] + idx, z);
		// printf("a = %.20f\n", a);
	}

	__global__ void
	kernel_calc_gL_2(
		int layer_id,
		int *l,
		int *s_ext,
		float *z_ext_arr,
		float *a_ext_arr,
		float *t_arr,
		float *gjl_ext
	 ){

		volatile int idx = threadIdx.x + blockDim.x*blockIdx.x;

		int neuron_count = l[layer_id];

		if(idx >= neuron_count-1) return;

		float z = z_ext_arr[s_ext[layer_id] + idx];
		float tmp = 1 + expf(-z);
		float f_deriv=expf(-z) / (tmp*tmp);

		gjl_ext[s_ext[layer_id] + idx] = f_deriv*(a_ext_arr[s_ext[layer_id] + idx] - t_arr[idx]);
	}

	__global__ void
	kernel_calc_gjL_2(
		int layer_id,
		int *l,
		int *s_ext,
		int *sw_ext,
		float *z_ext_arr,
		float *a_ext_arr,
		float *t_arr,
		float *gjl_ext,
		float *w_arr
	 ){

		volatile int idx = threadIdx.x + blockDim.x*blockIdx.x;

		int neuron_count = l[layer_id];
		int neuron_count_next = l[layer_id+1];

		if(idx >= neuron_count-1) return;

		//float f_deriv=expf(-z_arr[s[layer_id] + idx]) / powf((1 + expf(-z_arr[s[layer_id] + idx])),2.0f);
		float z = z_ext_arr[s_ext[layer_id] + idx];
		float tmp = 1 + expf(-z);
		float f_deriv=expf(-z) / (tmp*tmp);


		float sum = 0;
		for (int k = 0; k < neuron_count_next-1; k++) {
				sum += w_arr[sw_ext[layer_id] + idx*(l[layer_id + 1] - 1) + k] * gjl_ext[s_ext[layer_id + 1] + k];
		}

		gjl_ext[s_ext[layer_id] + idx] = f_deriv*sum;
		// printf("Kernelis %d - %.20f\n", s[layer_id] + idx, gjl[s[layer_id] + idx]);
	}


	__global__ void
	kernel_weight_update_2(
		int layer_id,
		int *l,
		int *s_ext,
		int *sw_ext,
		float *z_ext_arr,
		float *a_ext_arr,
		float *t_arr,
		float *gjl_ext,
		float *w_arr,
		float *dw_arr,
		float eta,
		float alpha
	 ){

		 volatile int idx = threadIdx.x + blockDim.x*blockIdx.x;

		 int neuron_count = l[layer_id];
		 int neuron_count_next = l[layer_id+1];

		 if(idx >= neuron_count) return;

		 float a = a_ext_arr[s_ext[layer_id] + idx];
		 for(int k = 0; k < neuron_count_next-1; k++){

			 float grad=/*a_arr[s[layer_id] + idx]*/a*gjl_ext[s_ext[layer_id + 1] + k];

			 dw_arr[sw_ext[layer_id] + idx*(neuron_count_next - 1) + k]=
					-eta*grad+
					alpha*dw_arr[sw_ext[layer_id] + idx*(neuron_count_next - 1) + k];

			 w_arr[sw_ext[layer_id] + idx*(neuron_count_next - 1) + k]+=
					dw_arr[sw_ext[layer_id] + idx*(neuron_count_next - 1) + k];
		 }
	}

}

void run_cuda_sample(){

  int deviceCount = 0;
  checkCudaErrors( hipGetDeviceCount(&deviceCount));
  if(deviceCount == 0){
    printf("*** there is no CUDE device\n");
    return;
  }

  checkCudaErrors( hipSetDevice(0) );

  int n = 11; // number of elements

  float *arr = new float[n];
  for(int i = 0; i < n; i++)
    arr[i] = i;

  int h = 4; // number of threads in block
  int g = (n + (h-n%h))/h; // number of grids

  printf("n=%d, h=%d, g=%d\n", n, h, g);


  int bc_arr = sizeof(float)*n;

  float *dv_arr = NULL;

  checkCudaErrors( hipMalloc((void **)&dv_arr, bc_arr) );

  checkCudaErrors( hipMemcpy(dv_arr, arr, bc_arr, hipMemcpyHostToDevice) );

  dim3 grid_dim(g, 1, 1);
  dim3 block_dim(h, 1, 1);

  ann::kernel<<<grid_dim, block_dim>>>(n, dv_arr);



  checkCudaErrors( hipMemcpy(arr, dv_arr, bc_arr, hipMemcpyDeviceToHost) );

  for(int i = 0; i < n; i++)
    printf("[%d] = %f\n", i, arr[i]);

  checkCudaErrors( hipFree(dv_arr) );

  checkCudaErrors(hipDeviceReset());

}



//
//AnnSerialFLT
//
void AnnCUDA::prepare( Topology *top){
	cTopology = top;

	l = new int[top->getLayerCount()];
	s = new int[top->getLayerCount()];

	int neuronCount = cTopology->obtainNeuronCount();
	int weightCount = cTopology->obtainWeightCount();

	a_arr = new float[neuronCount];
	z_arr = new float[neuronCount];

	W = new int[top->getLayerCount()];
	sw = new int[top->getLayerCount()];

	w_arr = new float[weightCount];
	dw_arr = new float[weightCount];

	t_arr = new float[top->getLayerSize(top->getLayerCount() - 1)];

	gjl = new float[neuronCount];

	// cuda

	int deviceCount = 0;
	checkCudaErrors( hipGetDeviceCount(&deviceCount));
	if(deviceCount == 0){
		printf("*** there is no CUDE device\n");
		return;
	}

	checkCudaErrors( hipSetDevice(0) );

	dv_l = NULL; bc_l = sizeof(int)*top->getLayerCount();
	dv_s = NULL; bc_s = sizeof(int)*top->getLayerCount();;

	dv_a_arr = NULL; bc_a_arr = sizeof(float)*neuronCount;
	dv_z_arr = NULL; bc_z_arr = sizeof(float)*neuronCount;

	dv_W = NULL; bc_W = sizeof(int)*top->getLayerCount();
	dv_sw = NULL; bc_sw = sizeof(int)*top->getLayerCount();

	dv_w_arr = NULL; bc_w_arr = sizeof(float)*weightCount;
	dv_dw_arr = NULL; bc_dw_arr = sizeof(float)*weightCount;

	dv_t_arr = NULL; bc_t_arr = sizeof(float)*top->getLayerSize(top->getLayerCount() - 1);
	dv_gjl = NULL; bc_gjl = sizeof(float)*neuronCount;

	checkCudaErrors( hipMalloc((void **)&dv_l, bc_l) );
	checkCudaErrors( hipMalloc((void **)&dv_s, bc_s) );
	checkCudaErrors( hipMalloc((void **)&dv_a_arr, bc_a_arr) );
	checkCudaErrors( hipMalloc((void **)&dv_z_arr, bc_z_arr) );
	checkCudaErrors( hipMalloc((void **)&dv_W, bc_W) );
	checkCudaErrors( hipMalloc((void **)&dv_sw, bc_sw) );
	checkCudaErrors( hipMalloc((void **)&dv_w_arr, bc_w_arr) );
	checkCudaErrors( hipMalloc((void **)&dv_dw_arr, bc_dw_arr) );
	checkCudaErrors( hipMalloc((void **)&dv_t_arr, bc_t_arr) );
	checkCudaErrors( hipMalloc((void **)&dv_gjl, bc_gjl) );

}

void AnnCUDA::init(FILE *pFile=NULL){
  L = cTopology->getLayerCount();

	Random *rnd = new Random();

	//Neuronu kiekiai sluoksnyje
	for (int i = 0; i < L; i++) {
		l[i] = cTopology->getLayerSize(i) + 1;
	}

	//Sluoksniu pradzios indeksai
	for (int i = 0; i < L; i++) {
		s[i] = 0;
		for (int j = i; j > 0; j--) {
			s[i] += l[j - 1];
		}
	}

	//Bias neuronai
	for (int i = 0; i < L - 1; i++) {
		a_arr[s[i + 1] - 1] = 1;
	}


	//Svoriu kiekiai l-ame sluoksnyje
	for (int i = 0; i < L - 1; i++) {
		W[i] = l[i] * (l[i + 1] - 1);
		sw[i] = 0;
		if (i != 0) {
			for (int j = 0; j < i; j++) {
				sw[i] += W[j];
			}
		}
  }

  for (int i = 0; i < L - 1; i++)
    for (int j = 0; j < W[i]; j++) {
      w_arr[sw[i] + j] =(rnd->next()*2-1); // (double)rand() / double(RAND_MAX);
      dw_arr[sw[i] + j] = 0.0;
  }

	checkCudaErrors( hipMemcpy(dv_w_arr, w_arr, bc_w_arr, hipMemcpyHostToDevice) );
	checkCudaErrors( hipMemcpy(dv_dw_arr, dw_arr, bc_dw_arr, hipMemcpyHostToDevice) );

	checkCudaErrors( hipMemcpy(dv_l, l, bc_l, hipMemcpyHostToDevice) );
	checkCudaErrors( hipMemcpy(dv_s, s, bc_s, hipMemcpyHostToDevice) );
	checkCudaErrors( hipMemcpy(dv_sw, sw, bc_sw, hipMemcpyHostToDevice) );
}

void AnnCUDA::train(float *a, float *b, float alpha, float eta){
	for (int i = 0; i < cTopology->getLayerSize(0); i++) {
		a_arr[i] = a[i];
	}

	for (int j = 0; j < cTopology->obtainNeuronCount(); j++) {
		z_arr[j] = 0;
	}

	calc_feedForward();

	// for (int i = 0; i < 10; i++) {
	// 	printf("a[%d] = %.10f\n", i, a_arr[i]);
	// }

	// for (int i = 0; i < 7; i++) {
	// 	printf("a[%d] = %.10f\n", i, a_arr[i]);
	// }
	// printf("\n");
	// for (int i = 0; i < 7; i++) {
	// 	printf("z[%d] = %.10f\n", i, z_arr[i]);
	// }

	for (int i = 0; i < cTopology->getLayerSize(cTopology->getLayerCount() - 1); i++) {
		t_arr[i] = b[i];
	}

	calc_gjl();

	// //back propogation:
	// for (int i = 0; i <L - 1; i++) {//per sluoksnius
	// 	for (int j = 0; j < l[i]; j++) {//per neuronus
	// 		for (int k = 0; k < l[i + 1] - 1; k++) {//per kito sluoksnio neuronus
	// 			dw_arr[sw[i] + k + j*(l[i + 1] - 1)] = delta_w(w_gradient(i, j, k), dw_arr[sw[i] + k + j*(l[i + 1] - 1)], alpha, eta);
	// 			w_arr[sw[i] + k + j*(l[i + 1] - 1)] += dw_arr[sw[i] + k + j*(l[i + 1] - 1)];
	// 		}
	// 	}
	// }

//	checkCudaErrors( hipMemcpy(dv_a_arr, a_arr, bc_a_arr, hipMemcpyHostToDevice) );
	//checkCudaErrors( hipMemcpy(dv_gjl, gjl, bc_gjl, hipMemcpyHostToDevice) );
	//checkCudaErrors( hipMemcpy(dv_w_arr, w_arr, bc_w_arr, hipMemcpyHostToDevice) );


	for (int i = 0; i < L-1; i++) {//per sluoksnius einu+

		int neuron_count = l[i];
		int h = 32; // number of threads in block
		int g = (neuron_count + (h-neuron_count%h))/h; // number of grids
		dim3 grid_dim(g, 1, 1);
		dim3 block_dim(h, 1, 1);

		// printf("%s\n", "A masyvas");
		// for (int j = 0; j < 7; j++) {
		// 	printf("a[%d] = %.20f\n", j, a_arr[j]);
		// }

		ann::kernel_weight_update<<<grid_dim, block_dim>>>(
			i,
			dv_l,
			dv_s,
			dv_sw,
			dv_z_arr,
			dv_a_arr,
			dv_t_arr,
			dv_gjl,
			dv_w_arr,
			dv_dw_arr,
			eta,
			alpha
		);
	}

	hipError_t err = hipGetLastError();
	if (err != hipSuccess)
	    printf("Error: %s\n", hipGetErrorString(err));

}

void AnnCUDA::finishTraining(){
	checkCudaErrors( hipMemcpy(w_arr, dv_w_arr, bc_w_arr, hipMemcpyDeviceToHost) );
	checkCudaErrors( hipMemcpy(dw_arr, dv_dw_arr, bc_dw_arr, hipMemcpyDeviceToHost) );
}

void AnnCUDA::feedForward(float *a, float *b){
	for (int i = 0; i < cTopology->getLayerSize(0); i++) {
		a_arr[i] = a[i];
	}

	for (int j = 0; j < cTopology->obtainNeuronCount(); j++) {
		z_arr[j] = 0;
	}


	calc_feedForward();

	checkCudaErrors( hipMemcpy(a_arr, dv_a_arr, bc_a_arr, hipMemcpyDeviceToHost) );

	for (int i = 0; i<cTopology->getLayerSize(cTopology->getLayerCount() - 1); i++){
		b[i] = a_arr[s[L - 1] + i];
		//printf("b[%d] = %.10f\n", i, b[i]);
	}
}

void AnnCUDA::calc_feedForward(){

	checkCudaErrors( hipMemcpy(dv_a_arr, a_arr, bc_a_arr, hipMemcpyHostToDevice) );



	for (int i = 1; i < L; i++) {//per sluoksnius einu+

//	printf("current layer_id = %d\n", i);
		int neuron_count = l[i];
		int h = 32; // number of threads in block
	  int g = (neuron_count + (h-neuron_count%h))/h; // number of grids
		dim3 grid_dim(g, 1, 1);
		dim3 block_dim(h, 1, 1);

		ann::kernel_feedforward<<<grid_dim, block_dim>>>(
			i,
			dv_l,
			dv_s,
			dv_sw,
			dv_z_arr,
			dv_a_arr,
			dv_w_arr
		);

	}
}

void AnnCUDA::calc_gjl(){

	checkCudaErrors( hipMemcpy(dv_t_arr, t_arr, bc_t_arr, hipMemcpyHostToDevice) );


	// int last_layer_id=cTopology->getLayerCount()-1;
	int last_layer_id=L-1;
	int neuron_count = l[last_layer_id];
	int h = 32; // number of threads in block
	int g = (neuron_count + (h-neuron_count%h))/h; // number of grids
	dim3 grid_dim(g, 1, 1);
	dim3 block_dim(h, 1, 1);


	ann::kernel_calc_gL<<<grid_dim, block_dim>>>(
		last_layer_id,
		dv_l,
		dv_s,
		dv_z_arr,
		dv_a_arr,
		dv_t_arr,
		dv_gjl
	);

	//Cia nezinau, ar i >= 0, ar i >= 1
	for (int i = L - 2; i >= 1; i--) {
			neuron_count = l[i];
			h = 32; // number of threads in block
			g = (neuron_count + (h-neuron_count%h))/h; // number of grids
			dim3 grid_dim(g, 1, 1);
			dim3 block_dim(h, 1, 1);

			ann::kernel_calc_gjL<<<grid_dim, block_dim>>>(
				i,
				dv_l,
				dv_s,
				dv_sw,
				dv_z_arr,
				dv_a_arr,
				dv_t_arr,
				dv_gjl,
				dv_w_arr
			);
		}

//	checkCudaErrors( hipMemcpy(gjl, dv_gjl, bc_gjl, hipMemcpyDeviceToHost) );

}

float AnnCUDA::delta_w(float grad, float dw, float alpha, float eta) {
	return -eta*grad + alpha*dw;
}

float AnnCUDA::gL(float a, float z, float t) {
	float w = f_deriv(z) * (a - t);
	return w;
}

float AnnCUDA::f(float x) {
		//return atanf(x)/M_PI + 0.5;
	float y = 1 + exp(-x);
	return 1 / y;
}

float AnnCUDA::f_deriv(float x) {
	//return  1.0 / (1.0+ x*x);
	 return exp(-x) / pow((1 + exp(-x)), 2);
}

float AnnCUDA::w_gradient(int layer_id, int w_i, int w_j) {
	return a_arr[s[layer_id] + w_i] * gjl[s[layer_id + 1] + w_j];
}

float AnnCUDA::obtainError(float *b){
	checkCudaErrors( hipMemcpy(a_arr, dv_a_arr, bc_a_arr, hipMemcpyDeviceToHost) );
	float error = 0;
	for(int i = 0; i < l[L-1] - 1; i++){
		float tmp = b[i] - a_arr[s[L-1] + i];
		error += tmp*tmp;
		//printf("a_arr[%d] = %.10f\n", s[L-1] + i, a_arr[s[L-1] + i]);
	}
	return error;
}

void AnnCUDA::destroy(){
	delete[] l;
	l = NULL;
	delete[] s;
	s = NULL;

	delete[] a_arr;
	a_arr = NULL;
	delete[] z_arr;
	z_arr = NULL;

	delete[] W;
	W = NULL;
	delete[] sw;
	sw = NULL;

	delete[] w_arr;
	w_arr = NULL;
	delete[] dw_arr;
	dw_arr = NULL;

	delete[] t_arr;
	t_arr = NULL;

	delete[] gjl;
	gjl = NULL;



	checkCudaErrors( hipFree(dv_l) );
	checkCudaErrors( hipFree(dv_s) );
	checkCudaErrors( hipFree(dv_a_arr) );
	checkCudaErrors( hipFree(dv_z_arr) );
	checkCudaErrors( hipFree(dv_W) );
	checkCudaErrors( hipFree(dv_sw) );
	checkCudaErrors( hipFree(dv_w_arr) );
	checkCudaErrors( hipFree(dv_dw_arr) );
	checkCudaErrors( hipFree(dv_t_arr) );
	checkCudaErrors( hipFree(dv_gjl) );


  checkCudaErrors(hipDeviceReset());
}

float* AnnCUDA::getWeights(){
	return w_arr;
}


void AnnCUDA::print_out(){
  printf("z = %e\n", z_arr[s[L-1]+0]);
	printf("g = %e\n", gjl[s[L-1]+0]);

	for(int i = 0; i < l[L-2]; i++){
		if(i < l[L-2]) printf("[%d] z=%e, a=%e, w=%e, grad = %e\n", i, z_arr[s[L-2]+i], a_arr[s[L-2]+i], w_arr[sw[L-2] + i*(l[L-1]-1)], a_arr[s[L-2]+i]*gjl[s[L-1]+0]);
	}
}

void AnnCUDA::setWeights(float *t_w_arr) {
	w_arr=t_w_arr;
	checkCudaErrors( hipMemcpy(dv_w_arr, w_arr, bc_w_arr, hipMemcpyHostToDevice) );
}

void AnnCUDA::printf_Network(string filename){
  FILE * pFile;
  const char * c = filename.c_str();
  pFile = fopen(c, "wb");
  cTopology->printTopology(pFile);

  int weightCount = cTopology->obtainWeightCount();

  double *w_arr_dbl = new double[weightCount];
  double *dw_arr_dbl = new double[weightCount];
  for(int i = 0; i < weightCount; i++){
    w_arr_dbl[i] = (double)w_arr[i];
    dw_arr_dbl[i] = (double)dw_arr[i];
  }

  fwrite (w_arr_dbl , sizeof(double), weightCount, pFile);
  fwrite (dw_arr_dbl , sizeof(double), weightCount, pFile);
  fclose (pFile);
}


//******************Cuda 2.0***************************************

void AnnCUDA2::prepare( Topology *top){
	cTopology = top;


	l = new int[top->getLayerCount()];
	l_ext = new int[top->getLayerCount()];
	s_ext = new int[top->getLayerCount()];


	int neuronCount = cTopology->obtainNeuronCount();
	int weightCount = cTopology->obtainWeightCount();

	int neuronCount_ext = obtainNeuronCountExt(cTopology);
	int weightCount_ext = obtainWeightCountExt(cTopology);

	// printf("neuronCount = %d\n", neuronCount);
	// printf("neuronCount2 = %d\n", neuronCount2);
	// printf("weightCount = %d\n", weightCount);
	// printf("weightCount2 = %d\n", weightCount2);

	a_ext_arr = new float[neuronCount_ext];
	z_ext_arr = new float[neuronCount_ext];

	sw_ext = new int[top->getLayerCount()];


	w_arr = new float[weightCount_ext];
	dw_arr = new float[weightCount_ext];

	t_arr = new float[top->getLayerSize(top->getLayerCount() - 1)];

	gjl_ext = new float[neuronCount_ext];

	// cuda

	int deviceCount = 0;
	checkCudaErrors( hipGetDeviceCount(&deviceCount));
	if(deviceCount == 0){
		printf("*** there is no CUDE device\n");
		return;
	}

	checkCudaErrors( hipSetDevice(0) );

	dv_l = NULL; bc_l = sizeof(int)*top->getLayerCount();
	dv_s_ext = NULL; bc_s_ext = sizeof(int)*top->getLayerCount();;

	dv_a_ext_arr = NULL; bc_a_ext_arr = sizeof(float)*neuronCount_ext;
	dv_z_ext_arr = NULL; bc_z_ext_arr = sizeof(float)*neuronCount_ext;

	dv_sw_ext = NULL; bc_sw_ext = sizeof(int)*top->getLayerCount();

	dv_w_arr = NULL; bc_w_arr = sizeof(float)*weightCount_ext;
	dv_dw_arr = NULL; bc_dw_arr = sizeof(float)*weightCount_ext;

	dv_t_arr = NULL; bc_t_arr = sizeof(float)*top->getLayerSize(top->getLayerCount() - 1);
	dv_gjl_ext = NULL; bc_gjl_ext = sizeof(float)*neuronCount_ext;

	checkCudaErrors( hipMalloc((void **)&dv_l, bc_l) );
	checkCudaErrors( hipMalloc((void **)&dv_s_ext, bc_s_ext) );
	checkCudaErrors( hipMalloc((void **)&dv_a_ext_arr, bc_a_ext_arr) );
	checkCudaErrors( hipMalloc((void **)&dv_z_ext_arr, bc_z_ext_arr) );
	checkCudaErrors( hipMalloc((void **)&dv_sw_ext, bc_sw_ext) );
	checkCudaErrors( hipMalloc((void **)&dv_w_arr, bc_w_arr) );
	checkCudaErrors( hipMalloc((void **)&dv_dw_arr, bc_dw_arr) );
	checkCudaErrors( hipMalloc((void **)&dv_t_arr, bc_t_arr) );
	checkCudaErrors( hipMalloc((void **)&dv_gjl_ext, bc_gjl_ext) );

}

void AnnCUDA2::init(FILE *pFile=NULL){
  L = cTopology->getLayerCount();

	int *W = new int[L];
	int *W_ext = new int[L];


	Random *rnd = new Random();

	//Neuronu kiekiai sluoksnyje
	for (int i = 0; i < L; i++) {
		int neuron_count = cTopology -> getLayerSize(i) + 1;
		l[i] = neuron_count;
		l_ext[i] = neuron_count + (32 - neuron_count % 32);
	}

	//Sluoksniu pradzios indeksai
	for (int i = 0; i < L; i++) {
		s_ext[i] = 0;
		for (int j = i; j > 0; j--) {
			s_ext[i] += l_ext[j - 1];
		}
	}

	//Bias neuronai
	for (int i = 0; i < L - 1; i++) {
		a_ext_arr[s_ext[i] + l[i] - 1] = 1;
	}

	//Svoriu kiekiai l-ame sluoksnyje
	for (int i = 0; i < L - 1; i++) {

		W[i] = l[i] * (l[i + 1] - 1);
		W_ext[i] = 	W[i];
		if (W_ext[i] % 32 != 0) {
			W_ext[i] += (32 - W_ext[i] % 32);
		}
		sw_ext[i] = 0;
		if (i != 0) {
			for (int j = 0; j < i; j++) {
				sw_ext[i] += W_ext[j];

			}
		}
  }

  for (int i = 0; i < L - 1; i++)
    for (int j = 0; j < W_ext[i]; j++) {
			if (j < W[i]){
      	w_arr[sw_ext[i] + j] =(rnd->next()*2-1);
			}
			else{
				w_arr[sw_ext[i] + j] = 0.0;
			}
      dw_arr[sw_ext[i] + j] = 0.0;
  }

	delete [] W;
	delete [] W_ext;

	checkCudaErrors( hipMemcpy(dv_w_arr, w_arr, bc_w_arr, hipMemcpyHostToDevice) );
	checkCudaErrors( hipMemcpy(dv_dw_arr, dw_arr, bc_dw_arr, hipMemcpyHostToDevice) );

	checkCudaErrors( hipMemcpy(dv_l, l, bc_l, hipMemcpyHostToDevice) );
	checkCudaErrors( hipMemcpy(dv_s_ext, s_ext, bc_s_ext, hipMemcpyHostToDevice) );
	checkCudaErrors( hipMemcpy(dv_sw_ext, sw_ext, bc_sw_ext, hipMemcpyHostToDevice) );
}

void AnnCUDA2::train(float *a, float *b, float alpha, float eta){
	for (int i = 0; i < cTopology->getLayerSize(0); i++) {
		a_ext_arr[i] = a[i];
	}

	for (int j = 0; j < obtainNeuronCountExt(cTopology); j++) {
		z_ext_arr[j] = 0;
	}

	calc_feedForward();

	for (int i = 0; i < cTopology->getLayerSize(cTopology->getLayerCount() - 1); i++) {
		t_arr[i] = b[i];
	}

	calc_gjl();

	for (int i = 0; i < L-1; i++) {//per sluoksnius einu+

		int neuron_count = l[i];
		int h = 32; // number of threads in block
		int g = (neuron_count + (h-neuron_count%h))/h; // number of grids
		dim3 grid_dim(g, 1, 1);
		dim3 block_dim(h, 1, 1);

		ann::kernel_weight_update_2<<<grid_dim, block_dim>>>(
			i,
			dv_l,
			dv_s_ext,
			dv_sw_ext,
			dv_z_ext_arr,
			dv_a_ext_arr,
			dv_t_arr,
			dv_gjl_ext,
			dv_w_arr,
			dv_dw_arr,
			eta,
			alpha
		);
	}

	hipError_t err = hipGetLastError();
	if (err != hipSuccess)
	    printf("Error: %s\n", hipGetErrorString(err));

}

void AnnCUDA2::finishTraining(){
	checkCudaErrors( hipMemcpy(w_arr, dv_w_arr, bc_w_arr, hipMemcpyDeviceToHost) );
	checkCudaErrors( hipMemcpy(dw_arr, dv_dw_arr, bc_dw_arr, hipMemcpyDeviceToHost) );
}

void AnnCUDA2::feedForward(float *a, float *b){
	for (int i = 0; i < cTopology->getLayerSize(0); i++) {
		a_ext_arr[i] = a[i];
	}

	for (int j = 0; j < obtainNeuronCountExt(cTopology); j++) {
		z_ext_arr[j] = 0;
	}

	calc_feedForward();

	checkCudaErrors( hipMemcpy(a_ext_arr, dv_a_ext_arr, bc_a_ext_arr, hipMemcpyDeviceToHost) );

	for (int i = 0; i < l[L - 1]; i++){
		b[i] = a_ext_arr[s_ext[L - 1] + i];
		//printf("b[%d] = %.10f\n", i, b[i]);
	}
}

void AnnCUDA2::calc_feedForward(){

	checkCudaErrors( hipMemcpy(dv_a_ext_arr, a_ext_arr, bc_a_ext_arr, hipMemcpyHostToDevice) );



	for (int i = 1; i < L; i++) {//per sluoksnius einu+

	//printf("current layer_id = %d\n", i);
		int neuron_count = l[i];
		int h = 32; // number of threads in block
	  int g = (neuron_count + (h-neuron_count%h))/h; // number of grids
		dim3 grid_dim(g, 1, 1);
		dim3 block_dim(h, 1, 1);

		ann::kernel_feedforward_2<<<grid_dim, block_dim>>>(
			i,
			dv_l,
			dv_s_ext,
			dv_sw_ext,
			dv_z_ext_arr,
			dv_a_ext_arr,
			dv_w_arr
		);
	}
}

void AnnCUDA2::calc_gjl(){

	checkCudaErrors( hipMemcpy(dv_t_arr, t_arr, bc_t_arr, hipMemcpyHostToDevice) );


	// int last_layer_id=cTopology->getLayerCount()-1;
	int last_layer_id = L-1;
	int neuron_count = l[last_layer_id];
	int h = 32; // number of threads in block
	int g = (neuron_count + (h-neuron_count%h))/h; // number of grids
	dim3 grid_dim(g, 1, 1);
	dim3 block_dim(h, 1, 1);


	ann::kernel_calc_gL_2<<<grid_dim, block_dim>>>(
		last_layer_id,
		dv_l,
		dv_s_ext,
		dv_z_ext_arr,
		dv_a_ext_arr,
		dv_t_arr,
		dv_gjl_ext
	);

	//Cia nezinau, ar i >= 0, ar i >= 1
	for (int i = L - 2; i >= 1; i--) {
			neuron_count = l[i];
			h = 32; // number of threads in block
			g = (neuron_count + (h-neuron_count%h))/h; // number of grids
			dim3 grid_dim(g, 1, 1);
			dim3 block_dim(h, 1, 1);

			ann::kernel_calc_gjL<<<grid_dim, block_dim>>>(
				i,
				dv_l,
				dv_s_ext,
				dv_sw_ext,
				dv_z_ext_arr,
				dv_a_ext_arr,
				dv_t_arr,
				dv_gjl_ext,
				dv_w_arr
			);
		}

		//	checkCudaErrors( hipMemcpy(gjl, dv_gjl, bc_gjl, hipMemcpyDeviceToHost) );

}

float AnnCUDA2::obtainError(float *b){
	checkCudaErrors( hipMemcpy(a_ext_arr, dv_a_ext_arr, bc_a_ext_arr, hipMemcpyDeviceToHost) );
	float error = 0;
	for(int i = 0; i < l[L-1] - 1; i++){
		float tmp = b[i] - a_ext_arr[s_ext[L-1] + i];
		error += tmp*tmp;
		//printf("a_arr[%d] = %.10f\n", s[L-1] + i, a_arr[s[L-1] + i]);
	}
	return error;
}

void AnnCUDA2::destroy(){
	delete[] l;
	l = NULL;

	delete[] l_ext;
	l_ext = NULL;


	delete[] s_ext;
	s_ext = NULL;

	delete[] a_ext_arr;
	a_ext_arr = NULL;
	delete[] z_ext_arr;
	z_ext_arr = NULL;

	delete[] sw_ext;
	sw_ext = NULL;

	delete[] w_arr;
	w_arr = NULL;
	delete[] dw_arr;
	dw_arr = NULL;

	delete[] t_arr;
	t_arr = NULL;

	delete[] gjl_ext;
	gjl_ext = NULL;



	checkCudaErrors( hipFree(dv_l) );
	checkCudaErrors( hipFree(dv_s_ext) );
	checkCudaErrors( hipFree(dv_a_ext_arr) );
	checkCudaErrors( hipFree(dv_z_ext_arr) );
	checkCudaErrors( hipFree(dv_sw_ext) );
	checkCudaErrors( hipFree(dv_w_arr) );
	checkCudaErrors( hipFree(dv_dw_arr) );
	checkCudaErrors( hipFree(dv_t_arr) );
	checkCudaErrors( hipFree(dv_gjl_ext) );


  checkCudaErrors(hipDeviceReset());
}

float* AnnCUDA2::getWeights(){
	return w_arr;
}

float* AnnCUDA2::getA(){
	return a_ext_arr;
}

void AnnCUDA2::print_out(){
  printf("z = %e\n", z_ext_arr[s_ext[L-1]+0]);
	printf("g = %e\n", gjl_ext[s_ext[L-1]+0]);

	for(int i = 0; i < l[L-2]; i++){
		if(i < l[L-2]) printf("[%d] z=%e, a=%e, w=%e, grad = %e\n",
		 	i, z_ext_arr[s_ext[L-2]+i],
			a_ext_arr[s_ext[L-2]+i],
		  w_arr[sw_ext[L-2] + i*(l[L-1]-1)],
			a_ext_arr[s_ext[L-2]+i]*gjl_ext[s_ext[L-1]+0]);
	}
}

void AnnCUDA2::setWeights(float *t_w_arr) {
	int prev_count = 0;
	for (int i = 0; i < cTopology->getLayerCount() - 1; i++) {

		for (int j = 0; j < l[i]*(l[i+1]-1); j++) {
			int index_w = sw_ext[i] + j;
			int index_t = prev_count + j;
			w_arr[index_w] = t_w_arr[index_t];
		}
		prev_count += l[i]*(l[i+1]-1);

	}

	checkCudaErrors( hipMemcpy(dv_w_arr, w_arr, bc_w_arr, hipMemcpyHostToDevice) );
}

void AnnCUDA2::printf_Network(string filename){
  FILE * pFile;
  const char * c = filename.c_str();
  pFile = fopen(c, "wb");
  cTopology->printTopology(pFile);

  int weightCount = cTopology->obtainWeightCount();

  double *w_arr_dbl = new double[weightCount];
  double *dw_arr_dbl = new double[weightCount];
	int sw_index = 0;
	for(int layer_id = 0; layer_id < L - 1; layer_id++){

		for(int k = 0; k < l[layer_id]*(l[layer_id+1]-1); k++){
			w_arr_dbl[sw_index+k] = (double)w_arr[sw_ext[layer_id]+k];
			dw_arr_dbl[sw_index+k] = (double)dw_arr[sw_ext[layer_id]+k];

		}
		sw_index +=  l[layer_id]*(l[layer_id+1]-1);
	}

  fwrite (w_arr_dbl , sizeof(double), weightCount, pFile);
  fwrite (dw_arr_dbl , sizeof(double), weightCount, pFile);
  fclose (pFile);
}

/* static */
int AnnCUDA2::obtainNeuronCountExt(Topology *top){
  int count = 0;
  for (int i = 0; i < top->getLayerCount(); i++){
    int neuron_count = top->getLayerSize(i)+1;
    count += neuron_count;
    if (neuron_count % 32 != 0)
      count += 32 - neuron_count % 32;
  }
  return count;
}

/* static */
int AnnCUDA2::obtainWeightCountExt(Topology *top){
  int count = 0;
  for (int i = 0; i < top->getLayerCount()-1; i++){
    int weight_count =  (top->getLayerSize(i)+1)*top->getLayerSize(i+1); //((*ml)[i] + 1)*(*ml)[i+1];
    count += weight_count;
    if (weight_count % 32 != 0)
      count += 32 - weight_count % 32;
  }
  return count;
}
